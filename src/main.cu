#include <iostream>
#include "CLI11.hpp"

#include <hip/hip_runtime.h>
#include <npp.h>

#include "UtilNPP/ImageIO.h"
#include "UtilNPP/ImagesCPU.h"
#include "UtilNPP/ImagesNPP.h"

#include "cudaAtScaleFinalAssignment/ImageTransformation.hpp"


int main(int argc, char** argv) {
    std::cout << "Hello World" << std::endl;
    ImageTransformation imgTrans;
    
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        std::cerr << "No CUDA capable devices found!" << std::endl;
        return 1;
    }
    std::cout << "Device count is: " << deviceCount << std::endl;
    hipSetDevice(0);

    npp::ImageCPU_8u_C1 oHostSrc;
    npp::ImageNPP_8u_C1 oDeviceSrc(oHostSrc);
 
     // create struct with box-filter mask size
     NppiSize oMaskSize = {5, 5};
 
     NppiSize oSrcSize = {(int)oDeviceSrc.width(), (int)oDeviceSrc.height()};
     NppiPoint oSrcOffset = {0, 0};
 
     // create struct with ROI size
     NppiSize oSizeROI = {(int)oDeviceSrc.width(), (int)oDeviceSrc.height()};
     // allocate device image of appropriately reduced size
     npp::ImageNPP_8u_C1 oDeviceDst(oSizeROI.width, oSizeROI.height);

    return EXIT_SUCCESS;
}